
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

#include <cstdlib> // for atoi

#include <fstream>

#define GRID_CENTER_OFFSET .25
#define GRID_SIZE 100



__global__ 
void rayTrace(int N, float *real_x, float *real_y, float *real_z, float timesteps){//, curandState *state){
    int rayIndex = blockIdx.x*blockDim.x + threadIdx.x;

    if(rayIndex < N){
        real_x[rayIndex] = ((float)rayIndex)/N*.5*GRID_CENTER_OFFSET*GRID_SIZE * cospi(60*((float)rayIndex)/N);
        real_y[rayIndex] = 0;
        real_z[rayIndex] = ((float)rayIndex)/N*.5*GRID_CENTER_OFFSET*GRID_SIZE * sinpi(60*((float)rayIndex)/N);
    }

    real_y[rayIndex] += timesteps;
}

int main(int argc, char *argv[]) 
{
    const int N = pow(GRID_SIZE, 3);

    int timesteps = 0;

    if(argc == 2){
        timesteps = atoi(argv[1]);
    }else{
        timesteps = 0;
    }

    
    float *real_x, *real_y, *real_z,*dev_real_x, *dev_real_y, *dev_real_z;
    
    real_x = (float*)malloc(N * sizeof(float));
    real_y = (float*)malloc(N * sizeof(float));
    real_z = (float*)malloc(N * sizeof(float));
    
    hipMalloc(&dev_real_x, N * sizeof(float));
    hipMalloc(&dev_real_y, N * sizeof(float));
    hipMalloc(&dev_real_z, N * sizeof(float));
    
    hipMemset(dev_real_x, 0, N * sizeof(float));
    hipMemset(dev_real_y, 0, N * sizeof(float));
    hipMemset(dev_real_z, 0, N * sizeof(float));

    rayTrace<<<(N+192)/256,256>>>(N, dev_real_x, dev_real_y, dev_real_z, timesteps);//, devStates);

    hipMemcpy(real_x, dev_real_x, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(real_y, dev_real_y, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(real_z, dev_real_z, N * sizeof(float), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    std::ofstream outputWrite;
    outputWrite.open("frames/output_"+std::to_string(timesteps)+".csv");

    std::cout<<"Generating until timestep: "<<timesteps<<std::endl;
    for (int i = 0; i < N; i++) {
        outputWrite << real_x[i] << "," << real_y[i] << "," << real_z[i]<<std::endl;
    }

    outputWrite.close();

    hipFree(dev_real_z);
    hipFree(dev_real_y);
    hipFree(dev_real_x);


}