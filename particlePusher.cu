
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

#include <cstdlib> // for atoi

#include <fstream>

#include <hiprand/hiprand_kernel.h>

#define GRID_CENTER_OFFSET .25
#define GRID_SIZE 100

// __global__
// void saxpy(int n, float a, float *x, float *y)
// {
//     int i = blockIdx.x*blockDim.x + threadIdx.x;
//     if (i < n) y[i] = a*x[i] + y[i];
// }

// __device__ 
// void initializeParticlePosition(int n, float *real_x, float *real_y, float *real_z){


// }

__global__ void setup_kernel(hiprandState *state)
{
    int id =  blockIdx.x*blockDim.x + threadIdx.x;
    /* Each thread gets same seed , a different sequence
    number , no offset */
    hiprand_init (1234 , id, 0, &state[id]);
}

__global__ void generate_kernel(hiprandState *state ,int *result){
    int id = threadIdx.x + blockIdx.x * 64;
    int count = 0;
    unsigned int x;
    /* Copy state to local memory for efficiency */
    hiprandState localState = state[id];
    /* Generate pseudo -random unsigned ints */
    for(int n = 0; n < 100000; n++) {
        x = hiprand (& localState);
        /* Check if low bit set */
        if(x & 1) {
            count ++;
        }
    }
    /* Copy state back to global memory */
    state[id] = localState;
    /* Store results */
    result[id] += count;
}

__global__ 
void rayTrace(int N, float *real_x, float *real_y, float *real_z, float timesteps, hiprandState *state){
    int rayIndex = blockIdx.x*blockDim.x + threadIdx.x;
    hiprandState localState = state[rayIndex];
    float random = hiprand_uniform(&localState);
    printf("%f \n--\n",random);
    if(rayIndex < N){
        real_x[rayIndex] = (GRID_SIZE/2 + (GRID_CENTER_OFFSET * GRID_SIZE * cospi(20 * ((float)rayIndex)/N)))* random*10000;
        real_y[rayIndex] = 0;
        real_z[rayIndex] = (GRID_SIZE/2 + (GRID_CENTER_OFFSET * GRID_SIZE * sinpi(20 * ((float)rayIndex)/N)))* random*10000;
    }
    // state[rayIndex] = localState;

    for(int t = 0; t < timesteps; t++){
        real_y[rayIndex] += 1;
    }
}

int main(int argc, char *argv[]) 
{
    const int N = pow(GRID_SIZE, 3);

    int timesteps = 0;

    if(argc == 2){
        timesteps = atoi(argv[1]);
    }else{
        timesteps = 0;
    }

    
    float *real_x, *real_y, *real_z,*dev_real_x, *dev_real_y, *dev_real_z;
    
    real_x = (float*)malloc(N * sizeof(float));
    real_y = (float*)malloc(N * sizeof(float));
    real_z = (float*)malloc(N * sizeof(float));
    
    hipMalloc(&dev_real_x, N * sizeof(float));
    hipMalloc(&dev_real_y, N * sizeof(float));
    hipMalloc(&dev_real_z, N * sizeof(float));
    
    hipMemset(dev_real_x, 0, N * sizeof(float));
    hipMemset(dev_real_y, 0, N * sizeof(float));
    hipMemset(dev_real_z, 0, N * sizeof(float));
    // cuRAND setup
    hiprandState *devStates;
    hipMalloc(&devStates, 2 * N * sizeof(float));

    setup_kernel<<<(2*N+128)/256,256>>>(devStates);
    hipDeviceSynchronize();

    rayTrace<<<(N+192)/256,256>>>(N, dev_real_x, dev_real_y, dev_real_z, timesteps, devStates);

    hipMemcpy(real_x, dev_real_x, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(real_y, dev_real_y, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(real_z, dev_real_z, N * sizeof(float), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    std::ofstream outputWrite;
    outputWrite.open("frames/output_"+std::to_string(timesteps)+".csv");

    std::cout<<"Generating until timestep: "<<timesteps<<std::endl;
    for (int i = 0; i < N; i++) {
        outputWrite << real_x[i] << "," << real_y[i] << "," << real_z[i]<<std::endl;
    }

    outputWrite.close();

    hipFree(devStates);
    hipFree(dev_real_z);
    hipFree(dev_real_y);
    hipFree(dev_real_x);


}