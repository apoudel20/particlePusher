
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

#include <cstdlib> // for atoi

#include <fstream>

#define GRID_CENTER_OFFSET .25
#define GRID_SIZE 100
typedef struct
{
    double* x;
    double* y;
    double* z;
}TimeLog;
TimeLog* new_cuda_TimeLog(int size)
{
    TimeLog* result = (TimeLog*)malloc(sizeof(TimeLog));
    printf("%p %p %p\n", &(result->x), &(result->y), &(result->z));
    hipMalloc(&(result->x), sizeof(double)*size);
    hipMalloc(&(result->y), sizeof(double)*size);
    hipMalloc(&(result->z), sizeof(double)*size);
    return result;
}
void copy_from_device(TimeLog* target, double* host_x,
                                       double* host_y,
                                       double* host_z,
                                       int size)
{
    if(target == NULL)
    {
        return;
    }
    hipMemcpy(host_x, target->x, sizeof(double)*size, hipMemcpyDeviceToHost);
    hipMemcpy(host_y, target->y, sizeof(double)*size, hipMemcpyDeviceToHost);
    hipMemcpy(host_z, target->z, sizeof(double)*size, hipMemcpyDeviceToHost);
}
__device__ float vectorFieldX(float real_x, float real_y, float real_z,float vel_x, float vel_y, float vel_z,int timesteps){
    float new_vel_x = real_x/10;//pow(real_x,2) * real_y;
    return new_vel_x;
}
__device__ float vectorFieldY(float real_x, float real_y, float real_z,float vel_x, float vel_y, float vel_z,int timesteps){
    float new_vel_y = 1/(exp(pow(0.1*real_x,4)))*1/(exp(pow(0.1*real_z,3)));//1/sqrt(1+pow((real_x),2) * pow((real_z),2));
    return new_vel_y;
}
__device__ float vectorFieldZ(float real_x, float real_y, float real_z,float vel_x, float vel_y, float vel_z,int timesteps){
    float new_vel_z = real_z/10;//real_x * real_y * real_z;
    return new_vel_z;
}
#define LOG 1
__global__ 
void rayTrace(int N, float *real_x, float *real_y, float *real_z,float *vel_x, float *vel_y, float *vel_z, int step, int timesteps, double dt, TimeLog* logger){//, curandState *state){
    int rayIndex = blockIdx.x*blockDim.x + threadIdx.x;
    if(rayIndex >= N)
    {
        return;
    }
    double *logx, *logy, *logz;
    if(LOG)
    {
        int offset = timesteps*rayIndex + step;
        logx = logger->x + offset;
        logy = logger->y + offset;
        logz = logger->z + offset;
    }

    double this_x, this_y, this_z;
    this_x = real_x[rayIndex];
    this_y = real_y[rayIndex];
    this_z = real_z[rayIndex];

    float d_vel_x, d_vel_y, d_vel_z;
    double this_vx, this_vy, this_vz; // intermediate values before calculating boundary conditions for reflection off walls

    this_vx = vel_x[rayIndex];
    this_vy = vel_y[rayIndex];
    this_vz = vel_z[rayIndex];
    d_vel_x = vectorFieldX(this_x, this_y, this_z, this_vx, this_vy, this_vz, step);
    d_vel_y = vectorFieldY(this_x, this_y, this_z, this_vx, this_vy, this_vz, step);
    d_vel_z = vectorFieldZ(this_x, this_y, this_z, this_vx, this_vy, this_vz, step); 
    this_vx = this_vx + d_vel_x;// + real_x[rayIndex];
    this_vy = this_vy + d_vel_y;// + real_y[rayIndex];
    this_vz = this_vz + d_vel_z;// + real_z[rayIndex];
    double future_x, future_y, future_z;
    future_x = this_vx*dt + this_x;
    future_y = this_vy*dt + this_y;
    future_z = this_vz*dt + this_z;
    // boundary reflection conditions
    float boundx = GRID_SIZE*.125;
    float boundy = GRID_SIZE*.08;
    float boundz = GRID_SIZE*.125;
    int condx = (future_x > boundx || future_x < -1*boundx);
    int condy = (future_y > boundy || future_y < -1*boundy);
    int condz = (future_z > boundz || future_z < -1*boundz);
    this_vx = -1*this_vx*condx + this_vx*!condx;
    this_vy = -1*this_vy*condy + this_vy*!condy;
    this_vz = -1*this_vz*condz + this_vz*!condz;
    this_x += this_vx*dt;
    this_y += this_vy*dt;
    this_z += this_vz*dt;
    if(LOG)
    {
        *logx = this_x;
        *logy = this_y;
        *logz = this_z;
    }
    real_x[rayIndex] = this_x;
    real_y[rayIndex] = this_y;
    real_z[rayIndex] = this_z;
    vel_x[rayIndex] = this_vx;
    vel_y[rayIndex] = this_vy;
    vel_z[rayIndex] = this_vz;
}

int main(int argc, char *argv[]) 
{
    const int N = pow(GRID_SIZE, 3);

    int timesteps = 0;

    if(argc == 2){
        timesteps = atoi(argv[1]);
    }else{
        timesteps = 0;
    }

    
    float *real_x, *real_y, *real_z,*dev_real_x, *dev_real_y, *dev_real_z; // R3 coordinates to be mapped onto the gridspace 
    float *vel_x, *vel_y, *vel_z, *dev_vel_x, *dev_vel_y, *dev_vel_z; // current velocities for the particles

    // allocate space for position arrays    
    real_x = (float*)malloc(N * sizeof(float));
    real_y = (float*)malloc(N * sizeof(float));
    real_z = (float*)malloc(N * sizeof(float));
    
    // allocate space for velocity arrays
    vel_x = (float*)malloc(N * sizeof(float));
    vel_y = (float*)malloc(N * sizeof(float));
    vel_z = (float*)malloc(N * sizeof(float));
    for(int i = 0; i < N; i++)
    {
        real_x[i] = ((float)i)/N*.35*GRID_CENTER_OFFSET*GRID_SIZE*cospi(.8*GRID_SIZE*((float)i)/N);
        real_y[i] = 1;
        real_z[i] = ((float)i)/N*.35*GRID_CENTER_OFFSET*GRID_SIZE*sinpi(.8*GRID_SIZE*((float)i)/N);

    }
    // allocate device space for position arrays
    hipMalloc(&dev_real_x, N * sizeof(float));
    hipMalloc(&dev_real_y, N * sizeof(float));
    hipMalloc(&dev_real_z, N * sizeof(float));

    // allocate device space for velocity arrays
    hipMalloc(&dev_vel_x, N * sizeof(float));
    hipMalloc(&dev_vel_y, N * sizeof(float));
    hipMalloc(&dev_vel_z, N * sizeof(float));
    
    // initialize arrays to 0
    hipMemcpy(dev_real_x, real_x, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_real_y, real_y, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_real_z, real_z, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(dev_vel_x, 0, N * sizeof(float));
    hipMemset(dev_vel_y, 0, N * sizeof(float));
    hipMemset(dev_vel_z, 0, N * sizeof(float));
    TimeLog* logger = new_cuda_TimeLog(timesteps*N), *cu_logger;
    hipMalloc(&cu_logger, sizeof(TimeLog));
    hipMemcpy(cu_logger, logger, sizeof(TimeLog), hipMemcpyHostToDevice);
    for(int i = 0; i < timesteps; i++)
    {
        rayTrace<<<((int)pow(GRID_SIZE, 3) + ((256 - (int)pow(GRID_SIZE, 3) % 256) % 256))/256,256>>>(N, dev_real_x, dev_real_y, dev_real_z,dev_vel_x, dev_vel_y, dev_vel_z, i, timesteps, 1e-1, cu_logger);//, devStates);
        hipDeviceSynchronize();
    }
    double* rx = (double*)malloc(sizeof(double)*timesteps*N);
    double* ry = (double*)malloc(sizeof(double)*timesteps*N);
    double* rz = (double*)malloc(sizeof(double)*timesteps*N);
    copy_from_device(logger, rx, ry, rz, timesteps*N);

    // return results to host
    hipMemcpy(real_x, dev_real_x, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(real_y, dev_real_y, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(real_z, dev_real_z, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(vel_x, dev_vel_x, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(vel_y, dev_vel_y, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(vel_z, dev_vel_z, N * sizeof(float), hipMemcpyDeviceToHost);

    std::ofstream outputWrite;
    std::cout<<"Generating until timestep: "<<timesteps<<std::endl;
    for(int i = 0; i < timesteps; i++)
    {
        outputWrite.open("frames/output_"+std::to_string(i)+".csv");

        for (int j = 0; j < N; j++) {
            outputWrite << rx[j*timesteps+i] << "," << ry[j*timesteps+i] << "," << rz[j*timesteps+i]<<std::endl;
        }
        outputWrite.close();
        printf("frames/output_%s.csv\n", std::to_string(i).c_str());
    }


    hipFree(dev_vel_z);
    hipFree(dev_vel_y);
    hipFree(dev_vel_x);    
    hipFree(dev_real_z);
    hipFree(dev_real_y);
    hipFree(dev_real_x);
    //cudaFree(cu_logger);
    //free(logger);
    free(rx);
    free(ry);
    free(rz);
    free(vel_x);
    free(vel_y);
    free(vel_z);
    free(real_z);
    free(real_y);
    free(real_x);
    


}
