
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

#include <cstdlib> // for atoi

#include <fstream>

#define GRID_CENTER_OFFSET .25
#define GRID_SIZE 100

__device__ float vectorFieldX(float real_x, float real_y, float real_z,float vel_x, float vel_y, float vel_z,int timesteps){
    float new_vel_x = real_x/10;//pow(real_x,2) * real_y;
    return new_vel_x;
}
__device__ float vectorFieldY(float real_x, float real_y, float real_z,float vel_x, float vel_y, float vel_z,int timesteps){
    float new_vel_y = 1/(exp(pow(0.1*real_x,4)))*1/(exp(pow(0.1*real_z,4)));//1/sqrt(1+pow((real_x),2) * pow((real_z),2));
    return new_vel_y;
}
__device__ float vectorFieldZ(float real_x, float real_y, float real_z,float vel_x, float vel_y, float vel_z,int timesteps){
    float new_vel_z = real_z/10;//real_x * real_y * real_z;
    return new_vel_z;
}

__global__ 
void rayTrace(int N, float *real_x, float *real_y, float *real_z,float *vel_x, float *vel_y, float *vel_z, int timesteps){//, curandState *state){
    int rayIndex = blockIdx.x*blockDim.x + threadIdx.x;

    if(rayIndex < N){
        real_x[rayIndex] = ((float)rayIndex)/N*.5*GRID_CENTER_OFFSET*GRID_SIZE * cospi(60*((float)rayIndex)/N);
        real_y[rayIndex] = 0;
        real_z[rayIndex] = ((float)rayIndex)/N*.5*GRID_CENTER_OFFSET*GRID_SIZE * sinpi(60*((float)rayIndex)/N);
    }
    float d_vel_x, d_vel_y, d_vel_z;

    d_vel_x = vectorFieldX(real_x[rayIndex], real_y[rayIndex], real_z[rayIndex], vel_x[rayIndex],vel_y[rayIndex],vel_z[rayIndex],timesteps);
    d_vel_y = vectorFieldY(real_x[rayIndex], real_y[rayIndex], real_z[rayIndex], vel_x[rayIndex],vel_y[rayIndex],vel_z[rayIndex],timesteps);
    d_vel_z = vectorFieldZ(real_x[rayIndex], real_y[rayIndex], real_z[rayIndex], vel_x[rayIndex],vel_y[rayIndex],vel_z[rayIndex],timesteps);    

    float inter_vel_x, inter_vel_y, inter_vel_z; // intermediate values before calculating boundary conditions for reflection off walls

    inter_vel_x = vel_x[rayIndex] + d_vel_x + real_x[rayIndex];
    inter_vel_y = vel_y[rayIndex] + d_vel_y + real_y[rayIndex];
    inter_vel_z = vel_z[rayIndex] + d_vel_z + real_z[rayIndex];

    // boundary reflection conditions
    vel_x[rayIndex] += d_vel_x;
    if(inter_vel_x > GRID_SIZE/8){
        real_x[rayIndex] = 2 * GRID_SIZE/8 - inter_vel_x;
        vel_x[rayIndex] = -vel_x[rayIndex];
    }else if(inter_vel_x < -GRID_SIZE/8){
        real_x[rayIndex] = -GRID_SIZE/8 - inter_vel_x;
        vel_x[rayIndex] = -vel_x[rayIndex];
    }else{
        real_x[rayIndex] += vel_x[rayIndex];
    }

    vel_y[rayIndex] += d_vel_y;
    if(inter_vel_y > GRID_SIZE/10){
        real_y[rayIndex] = 2 * GRID_SIZE/10 - inter_vel_y;
        vel_y[rayIndex] = -vel_y[rayIndex];
    }else if(inter_vel_y < 0){
        real_y[rayIndex] = -inter_vel_y;
        vel_y[rayIndex] = -vel_y[rayIndex];
    }else{
        real_y[rayIndex] += vel_y[rayIndex];
    }

    vel_z[rayIndex] += d_vel_z;
    if(inter_vel_z > GRID_SIZE/8){
        real_z[rayIndex] = 2 * GRID_SIZE/8 - inter_vel_z;
        vel_z[rayIndex] = -vel_z[rayIndex];
    }else if(inter_vel_z < -GRID_SIZE/8){
        real_z[rayIndex] = -GRID_SIZE/8 - inter_vel_z;
        vel_z[rayIndex] = -vel_z[rayIndex];
    }else{
        real_z[rayIndex] += vel_z[rayIndex];
    }

}

int main(int argc, char *argv[]) 
{
    const int N = pow(GRID_SIZE, 3);

    int timesteps = 0;

    if(argc == 2){
        timesteps = atoi(argv[1]);
    }else{
        timesteps = 0;
    }

    
    float *real_x, *real_y, *real_z,*dev_real_x, *dev_real_y, *dev_real_z; // R3 coordinates to be mapped onto the gridspace 
    float *vel_x, *vel_y, *vel_z, *dev_vel_x, *dev_vel_y, *dev_vel_z; // current velocities for the particles

    // allocate space for position arrays    
    real_x = (float*)malloc(N * sizeof(float));
    real_y = (float*)malloc(N * sizeof(float));
    real_z = (float*)malloc(N * sizeof(float));
    
    // allocate space for velocity arrays
    vel_x = (float*)malloc(N * sizeof(float));
    vel_y = (float*)malloc(N * sizeof(float));
    vel_z = (float*)malloc(N * sizeof(float));

    // allocate device space for position arrays
    hipMalloc(&dev_real_x, N * sizeof(float));
    hipMalloc(&dev_real_y, N * sizeof(float));
    hipMalloc(&dev_real_z, N * sizeof(float));

    // allocate device space for velocity arrays
    hipMalloc(&dev_vel_x, N * sizeof(float));
    hipMalloc(&dev_vel_y, N * sizeof(float));
    hipMalloc(&dev_vel_z, N * sizeof(float));
    
    // initialize arrays to 0
    hipMemset(dev_real_x, 0, N * sizeof(float));
    hipMemset(dev_real_y, 0, N * sizeof(float));
    hipMemset(dev_real_z, 0, N * sizeof(float));
    hipMemset(dev_vel_x, 0, N * sizeof(float));
    hipMemset(dev_vel_y, 0, N * sizeof(float));
    hipMemset(dev_vel_z, 0, N * sizeof(float));

    for(int t = 0; t < timesteps; t++){
        rayTrace<<<(N+192)/256,256>>>(N, dev_real_x, dev_real_y, dev_real_z,dev_vel_x, dev_vel_y, dev_vel_z, t);//, devStates);
        hipDeviceSynchronize();
    }

    // return results to host
    hipMemcpy(real_x, dev_real_x, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(real_y, dev_real_y, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(real_z, dev_real_z, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(vel_x, dev_vel_x, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(vel_y, dev_vel_y, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(vel_z, dev_vel_z, N * sizeof(float), hipMemcpyDeviceToHost);


    std::ofstream outputWrite;
    outputWrite.open("frames/output_"+std::to_string(timesteps)+".csv");

    std::cout<<"Generating until timestep: "<<timesteps<<std::endl;
    for (int i = 0; i < N; i++) {
        outputWrite << real_x[i] << "," << real_y[i] << "," << real_z[i]<<std::endl;
    }

    outputWrite.close();

    hipFree(dev_vel_z);
    hipFree(dev_vel_y);
    hipFree(dev_vel_x);    
    hipFree(dev_real_z);
    hipFree(dev_real_y);
    hipFree(dev_real_x);

    free(vel_z);
    free(vel_y);
    free(vel_x);
    free(real_z);
    free(real_y);
    free(real_x);
    


}